
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <omp.h>

//Esse código  realiza a multiplicação de matrizes quadradas usando CUDA

#define N 16
#define M 16
#define P 16

__global__ void matrix_multiply(int* A, int* B, int* C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

        int sum = 0;
        for (int i = 0; i < M; i++) {
            sum += A[row * M + i] * B[i * P + col];
        }
        C[row * P + col] = sum;
}

void mostraMat(int *mat, int tam){
   int i, j;

   for(i = 0; i < N; i++){
      for(j = 0; j < P; j++){
         printf("%d ", mat[i * N + j]);
      }
      printf("\n");
   }

}

void formaMat(int *mat, int tam, int fator){
   int i, j;

   for(i = 0; i < N; i++){
      for(j = 0; j < P; j++){
         mat[i * P + j] = i + j + fator;
      }
   }

}

int main() {
    int* A;
    int* B;
    int* C;
    int* d_A;
    int* d_B;
    int* d_C;

    // Alocar memória para as matrizes na CPU
    A = (int*)malloc(N * M * sizeof(int));
    B = (int*)malloc(M * P * sizeof(int));
    C = (int*)malloc(N * P * sizeof(int));

    formaMat(A, M, 1);
    formaMat(B, M, 5);

    //mostraMat(A, N);
    //printf("\n----------------------------------------\n");
    //mostraMat(B, N);
    
    // Inicializar matrizes A e B com dados

    // Alocar memória para as matrizes na GPU
    hipMalloc((void**)&d_A, N * M * sizeof(int));
    hipMalloc((void**)&d_B, M * P * sizeof(int));
    hipMalloc((void**)&d_C, N * P * sizeof(int));

    // Copiar dados da CPU para a GPU
    hipMemcpy(d_A, A, N * M * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, M * P * sizeof(int), hipMemcpyHostToDevice);

    // Definir tamanho dos blocos e grid
    dim3 blockSize(16, 16);
    dim3 gridSize((P + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    // Chamar o kernel para a multiplicação de matrizes
    matrix_multiply<<<gridSize, blockSize>>>(d_A, d_B, d_C);

    // Copiar o resultado de volta da GPU para a CPU
    hipMemcpy(C, d_C, N * P * sizeof(int), hipMemcpyDeviceToHost);
    //formaMat(C, N, 10);
    mostraMat(C, P);
    // Liberar memória na GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Liberar memória na CPU
    free(A);
    free(B);
    free(C);

    return 0;
}
