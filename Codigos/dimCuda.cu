#include <stdio.h>
#include <hip/hip_runtime.h>

//Esse código exibe as dimensões maximas da placa de video,
// para a execução de operações em CUDA
int main() {
    int deviceID = 0;  // ID da GPU em uso
    hipDeviceProp_t deviceProp;

    // Obter propriedades do dispositivo
    hipGetDeviceProperties(&deviceProp, deviceID);

    printf("Dimensões máximas da grid:\n");
    printf("Eixo x: %d\n", deviceProp.maxGridSize[0]);
    printf("Eixo y: %d\n", deviceProp.maxGridSize[1]);
    printf("Eixo z: %d\n", deviceProp.maxGridSize[2]);

    return 0;
}
